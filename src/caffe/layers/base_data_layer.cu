#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

	template <typename Dtype>
	void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
		// Reshape to loaded data.
		top[0]->ReshapeLike(batch->data_);
		// Copy the data
		caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
			top[0]->mutable_gpu_data());
		if (this->output_labels_) {
			// Reshape to loaded labels.
			top[1]->ReshapeLike(batch->label_);
			// Copy the labels.
			caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
				top[1]->mutable_gpu_data());
		}
		// Ensure the copy is synchronous wrt the host, so that the next batch isn't
		// copied in meanwhile.
		CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
		prefetch_free_.push(batch);
	}

	template <typename Dtype>
	void ExBasePrefetchingDataLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		ExBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
		// Reshape to loaded data.
		top[0]->ReshapeLike(batch->data_);
		top[1]->ReshapeLike(batch->label_);
		top[2]->ReshapeLike(batch->dense_);
		// Copy the data
		caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
			top[0]->mutable_gpu_data());
		caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
			top[1]->mutable_gpu_data());
		caffe_copy(batch->dense_.count(), batch->dense_.gpu_data(),
			top[2]->mutable_gpu_data());
		// Ensure the copy is synchronous wrt the host, so that the next batch isn't
		// copied in meanwhile.
		CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
		prefetch_free_.push(batch);
	}

	template <typename Dtype>
	void RoiPrefetchingDataLayer<Dtype>::Forward_gpu(
		const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
		RoiBatch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
		// Reshape to loaded data.
		top[0]->ReshapeLike(batch->data_);
		top[1]->ReshapeLike(batch->info_);
		if (output_labels_)
		{
			top[2]->ReshapeLike(batch->label_a_);
			top[3]->ReshapeLike(batch->label_c_);
			top[4]->ReshapeLike(batch->label_s_);
		}
		// Copy the data
		caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
			top[0]->mutable_gpu_data());
		caffe_copy(batch->info_.count(), batch->info_.gpu_data(),
			top[1]->mutable_gpu_data());
		if (output_labels_) {
			caffe_copy(batch->label_a_.count(), batch->label_a_.gpu_data(),
				top[2]->mutable_gpu_data());
			caffe_copy(batch->label_c_.count(), batch->label_c_.gpu_data(),
				top[3]->mutable_gpu_data());
			caffe_copy(batch->label_s_.count(), batch->label_s_.gpu_data(),
				top[4]->mutable_gpu_data());
		}
		// Ensure the copy is synchronous wrt the host, so that the next batch isn't
		// copied in meanwhile.
		CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
		prefetch_free_.push(batch);
	}

	INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);
	INSTANTIATE_LAYER_GPU_FORWARD(ExBasePrefetchingDataLayer);
	INSTANTIATE_LAYER_GPU_FORWARD(RoiPrefetchingDataLayer);

}  // namespace caffe
