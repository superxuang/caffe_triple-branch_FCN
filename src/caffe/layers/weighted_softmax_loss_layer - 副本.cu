#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <glog/logging.h>

#include "caffe/layer.hpp"
#include "caffe/layers/weighted_softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {  

template <typename Dtype>
__global__ void CalculateChannelWeights(const int nthreads,
          const Dtype* label, const int num, const int channel_dim, 
		  const int spatial_dim, const bool has_ignore_label_, const int ignore_label_,
          int* weights) {
  extern __shared__ int tmp_weights[];
  if (threadIdx.x < num * channel_dim)
	tmp_weights[threadIdx.x] = 0;
  __syncthreads();
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (!has_ignore_label_ || label_value != ignore_label_) {
	  atomicAdd(&(tmp_weights[n * channel_dim + label_value]), 1);
    }
  }
  __syncthreads();
  if (threadIdx.x < num * channel_dim)
    atomicAdd(&(weights[threadIdx.x]), tmp_weights[threadIdx.x]);
}

template <typename Dtype>
__global__ void WeightedSoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
		  const Dtype* weights_data, const int num, const int dim, 
		  const int spatial_dim, const bool has_ignore_label_, 
		  const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -weights_data[n * channels + label_value] * log(max(prob_data[n * dim + label_value * spatial_dim + s], Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void WeightedSoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  Blob<int> tmp_weights;
  std::vector<int> weights_shape(2);
  weights_shape[0] = outer_num_;
  weights_shape[1] = channel_num_;
  tmp_weights.Reshape(weights_shape);
  caffe_memset(outer_num_ * channel_num_ * sizeof(int), 0, tmp_weights.mutable_cpu_data());
  int* tmp_weights_gpu_data = tmp_weights.mutable_gpu_data();
  CalculateChannelWeights<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS, outer_num_ * channel_num_ * sizeof(Dtype)>>>(nthreads, label, outer_num_, 
	  channel_num_, inner_num_, has_ignore_label_, ignore_label_, tmp_weights_gpu_data);
  int* tmp_weights_cpu_data = tmp_weights.mutable_cpu_data();
  Dtype* weights_cpu_data = channel_weights_.mutable_cpu_data();
  for (int i = 0; i < outer_num_; ++i) {
	Dtype max_weight = 0;
	for (int j = 0; j < channel_num_; ++j) {
	  if (tmp_weights_cpu_data[i * channel_num_ + j] > max_weight) {
		max_weight = tmp_weights_cpu_data[i * channel_num_ + j];
	  }
	  LOG(INFO) << "weights_data = " << tmp_weights_cpu_data[i * channel_num_ + j];
	}
	LOG(INFO) << "max_weight = " << max_weight;
	for (int j = 0; j < channel_num_; ++j) {
	  weights_cpu_data[i * channel_num_ + j] = 1;
	  if (tmp_weights_cpu_data[i * channel_num_ + j] > 0)
	    weights_cpu_data[i * channel_num_ + j] = max_weight / tmp_weights_cpu_data[i * channel_num_ + j];
	  else
	    weights_cpu_data[i * channel_num_ + j] = 0;
	  LOG(INFO) << "normalized_weights_data = " << weights_cpu_data[i * channel_num_ + j];
	}
  }
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  Dtype* weights_gpu_data = channel_weights_.mutable_gpu_data();
  WeightedSoftmaxLossForwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data, weights_gpu_data, 
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  Dtype valid_count = -1;
  // Only launch another CUDA kernel if we actually need the count of valid
  // outputs.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(nthreads, counts, &valid_count);
  }
  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
                                                        valid_count);
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void WeightedSoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, const Dtype* weights_data, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
	  for (int c = 0; c < channels; ++c) {  
        bottom_diff[n * dim + c * spatial_dim + s] *= weights_data[n * channels + label_value];  
      }
    }
  }
}

template <typename Dtype>
void WeightedSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
	const Dtype* weights_data = channel_weights_.gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
	WeightedSoftmaxLossBackwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, weights_data, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID &&
        has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    const Dtype loss_weight = top[0]->cpu_diff()[0] /
                              get_normalizer(normalization_, valid_count);
    caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(WeightedSoftmaxWithLossLayer);

}  // namespace caffe
