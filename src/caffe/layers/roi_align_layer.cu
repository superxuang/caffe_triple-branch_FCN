#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/roi_align_layer.hpp"


using std::max;
using std::min;

namespace caffe {

	template <typename Dtype>
	__global__ void ROIAlignForward(const int nthreads, const Dtype* bottom_data,
		const Dtype spatial_scale_xy, const Dtype spatial_scale_z, const int channels, 
		const int length, const int height, const int width, 
		const int pooled_length, const int pooled_height, const int pooled_width,
		const Dtype* bottom_rois, const int interpolate_times, Dtype* top_data, int* argmax_data, Dtype* w_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {

			// (n, c, pl, ph, pw) is an element in the pooled output
			int pw = index % pooled_width;
			int ph = (index / pooled_width) % pooled_height;
			int pl = (index / pooled_width / pooled_height) % pooled_length;
			int c = (index / pooled_width / pooled_height / pooled_length) % channels;
			int n = index / pooled_width / pooled_height / pooled_length / channels;

			bottom_rois += n * 7;
			int roi_batch_ind = bottom_rois[0];

			Dtype roi_start_w = bottom_rois[1] * spatial_scale_xy;
			Dtype roi_start_h = bottom_rois[2] * spatial_scale_xy;
			Dtype roi_start_l = bottom_rois[3] * spatial_scale_z;
			Dtype roi_end_w = bottom_rois[4] * spatial_scale_xy;
			Dtype roi_end_h = bottom_rois[5] * spatial_scale_xy;
			Dtype roi_end_l = bottom_rois[6] * spatial_scale_z;
			// clipping
			roi_start_w = max(roi_start_w, Dtype(0)); 
			roi_start_h = max(roi_start_h, Dtype(0));
			roi_start_l = max(roi_start_l, Dtype(0));
			int img_width = round(width / spatial_scale_xy);
			int img_height = round(height / spatial_scale_xy);
			int img_length = round(length / spatial_scale_z);
			roi_end_w = min(Dtype(img_width - 1), roi_end_w);
			roi_end_h = min(Dtype(img_height - 1), roi_end_h);
			roi_end_l = min(Dtype(img_length - 1), roi_end_l);

			Dtype roi_length = max(roi_end_l - roi_start_l + 1, Dtype(1));
			Dtype roi_height = max(roi_end_h - roi_start_h + 1, Dtype(1));
			Dtype roi_width = max(roi_end_w - roi_start_w + 1, Dtype(1));
			const Dtype bin_size_l = static_cast<Dtype>(roi_length)
				/ static_cast<Dtype>(roi_length);
			const Dtype bin_size_h = static_cast<Dtype>(roi_height)
				/ static_cast<Dtype>(pooled_height);
			const Dtype bin_size_w = static_cast<Dtype>(roi_width)
				/ static_cast<Dtype>(pooled_width);

			bottom_data += (roi_batch_ind * channels + c) * length * height * width;

			double argmax_temp_data[8];
			double w_temp_data[8];
			double start_x = 0.25, start_y = 0.25, start_z = 0.25;
			if (interpolate_times == 1) {
				start_x = 0.5;
				start_y = 0.5;
				start_z = 0.5;
			}
			Dtype dfValue = 0, maxValue = 0;
			for (int inter_index = 0; inter_index < interpolate_times; ++inter_index) {
				int index_x = inter_index / 4;
				int index_y = inter_index / 2;
				int index_z = inter_index % 2;
				Dtype off_x = index_x * 0.5 + start_x;
				Dtype off_y = index_y * 0.5 + start_y;
				Dtype off_z = index_z * 0.5 + start_z;
				Dtype lcenter = static_cast<Dtype>(pl + off_x)* bin_size_l;
				Dtype hcenter = static_cast<Dtype>(ph + off_y)* bin_size_h;
				Dtype wcenter = static_cast<Dtype>(pw + off_z)* bin_size_w;

				lcenter = min(max(lcenter + roi_start_l, Dtype(0)), Dtype(length - 1));
				hcenter = min(max(hcenter + roi_start_h, Dtype(0)), Dtype(height - 1));
				wcenter = min(max(wcenter + roi_start_w, Dtype(0)), Dtype(width - 1));

				int lstart = min(max(lcenter, Dtype(0)), Dtype(length - 1));
				int hstart = min(max(hcenter, Dtype(0)), Dtype(height - 1));
				int wstart = min(max(wcenter, Dtype(0)), Dtype(width - 1));
				int lend = min(max(lstart + 1, 0), length - 1);
				int hend = min(max(hstart + 1, 0), height - 1);
				int wend = min(max(wstart + 1, 0), width - 1);

				Dtype fX0 = wcenter - wstart;
				Dtype fX1 = wend - wcenter;
				Dtype fY0 = hcenter - hstart;
				Dtype fY1 = hend - hcenter;
				Dtype fZ0 = lcenter - lstart;
				Dtype fZ1 = lend - lcenter;
				Dtype fFactorA = fZ1 * fY1 * fX1;
				Dtype fFactorB = fZ1 * fY1 * fX0;
				Dtype fFactorC = fZ1 * fY0 * fX1;
				Dtype fFactorD = fZ1 * fY0 * fX0;
				Dtype fFactorE = fZ0 * fY1 * fX1;
				Dtype fFactorF = fZ0 * fY1 * fX0;
				Dtype fFactorG = fZ0 * fY0 * fX1;
				Dtype fFactorH = fZ0 * fY0 * fX0;

				dfValue = bottom_data[lstart * height * width + hstart * width + wstart] * fFactorA
					+ bottom_data[lstart * height * width + hstart * width + wend] * fFactorB
					+ bottom_data[lstart * height * width + hend * width + wstart] * fFactorC
					+ bottom_data[lstart * height * width + hend * width + wend] * fFactorD
					+ bottom_data[lend * height * width + hstart * width + wstart] * fFactorE
					+ bottom_data[lend * height * width + hstart * width + wend] * fFactorF
					+ bottom_data[lend * height * width + hend * width + wstart] * fFactorG
					+ bottom_data[lend * height * width + hend * width + wend] * fFactorH;

				if (inter_index == 0) {
					maxValue = dfValue - 1;
				}

				argmax_temp_data[0] = lstart * height * width + hstart * width + wstart;
				argmax_temp_data[1] = lstart * height * width + hstart * width + wend;
				argmax_temp_data[2] = lstart * height * width + hend * width + wstart;
				argmax_temp_data[3] = lstart * height * width + hend * width + wend;
				argmax_temp_data[4] = lend * height * width + hstart * width + wstart;
				argmax_temp_data[5] = lend * height * width + hstart * width + wend;
				argmax_temp_data[6] = lend * height * width + hend * width + wstart;
				argmax_temp_data[7] = lend * height * width + hend * width + wend;

				w_temp_data[0] = fFactorA;
				w_temp_data[1] = fFactorB;
				w_temp_data[2] = fFactorC;
				w_temp_data[3] = fFactorD;
				w_temp_data[4] = fFactorE;
				w_temp_data[5] = fFactorF;
				w_temp_data[6] = fFactorG;
				w_temp_data[7] = fFactorH;

				if (dfValue > maxValue || inter_index == 0) {
					maxValue = dfValue;
					top_data[index] = dfValue;
					for (int s = 0; s < 8; ++s) {
						w_data[8 * index + s] = w_temp_data[s];
						argmax_data[8 * index + s] = argmax_temp_data[s];
					}
				}
			}
		}
	}

	template <typename Dtype>
	__global__ void ROICubicForward(const int nthreads, const Dtype* bottom_data,
		const Dtype spatial_scale_xy, const Dtype spatial_scale_z, const int channels, 
		const int length, const int height, const int width,
		const int pooled_length, const int pooled_height, const int pooled_width,
		const Dtype* bottom_rois, const int interpolate_times, Dtype* top_data, int* argmax_data, Dtype* w_data) {
		CUDA_KERNEL_LOOP(index, nthreads) {

			// (n, c, pl, ph, pw) is an element in the pooled output
			int pw = index % pooled_width;
			int ph = (index / pooled_width) % pooled_height;
			int pl = (index / pooled_width / pooled_height) % pooled_length;
			int c = (index / pooled_width / pooled_height / pooled_length) % channels;
			int n = index / pooled_width / pooled_height / pooled_length / channels;

			bottom_rois += n * 7;
			int roi_batch_ind = bottom_rois[0];

			Dtype roi_start_w = bottom_rois[1] * spatial_scale_xy;
			Dtype roi_start_h = bottom_rois[2] * spatial_scale_xy;
			Dtype roi_start_l = bottom_rois[3] * spatial_scale_z;
			Dtype roi_end_w = bottom_rois[4] * spatial_scale_xy;
			Dtype roi_end_h = bottom_rois[5] * spatial_scale_xy;
			Dtype roi_end_l = bottom_rois[6] * spatial_scale_z;
			// clipping
			roi_start_w = max(roi_start_w, Dtype(0));
			roi_start_h = max(roi_start_h, Dtype(0));
			roi_start_l = max(roi_start_l, Dtype(0));
			int img_width = round(width / spatial_scale_xy);
			int img_height = round(height / spatial_scale_xy);
			int img_length = round(length / spatial_scale_z);
			roi_end_w = min(Dtype(img_width - 1), roi_end_w);
			roi_end_h = min(Dtype(img_height - 1), roi_end_h);
			roi_end_l = min(Dtype(img_length - 1), roi_end_l);

			Dtype roi_length = max(roi_end_l - roi_start_l + 1, Dtype(1));
			Dtype roi_height = max(roi_end_h - roi_start_h + 1, Dtype(1));
			Dtype roi_width = max(roi_end_w - roi_start_w + 1, Dtype(1));
			const Dtype bin_size_l = static_cast<Dtype>(roi_length)
				/ static_cast<Dtype>(roi_length);
			const Dtype bin_size_h = static_cast<Dtype>(roi_height)
				/ static_cast<Dtype>(pooled_height);
			const Dtype bin_size_w = static_cast<Dtype>(roi_width)
				/ static_cast<Dtype>(pooled_width);

			bottom_data += (roi_batch_ind * channels + c) * length * height * width;
			double argmax_temp_data[64];
			double w_temp_data[64];
			double start_x = 0.25, start_y = 0.25, start_z = 0.25;
			if (interpolate_times == 1) {
				start_x = 0.5;
				start_y = 0.5;
				start_z = 0.5;
			}
			Dtype dfCubicValue = 0, maxValue = 0;
			for (int inter_index = 0; inter_index < interpolate_times; ++inter_index) {
				int index_x = inter_index / 4;
				int index_y = inter_index / 2;
				int index_z = inter_index % 2;
				Dtype off_x = index_x * 0.5 + start_x;
				Dtype off_y = index_y * 0.5 + start_y;
				Dtype off_z = index_z * 0.5 + start_z;
				Dtype lcenter = static_cast<Dtype>(pl + off_x)* bin_size_l;
				Dtype hcenter = static_cast<Dtype>(ph + off_y)* bin_size_h;
				Dtype wcenter = static_cast<Dtype>(pw + off_z)* bin_size_w;

				lcenter = min(max(lcenter + roi_start_l, Dtype(0)), Dtype(length - 1));
				hcenter = min(max(hcenter + roi_start_h, Dtype(0)), Dtype(height - 1));
				wcenter = min(max(wcenter + roi_start_w, Dtype(0)), Dtype(width - 1));

				int i = wcenter;
				int j = hcenter;
				int k = lcenter;

				/*get adjacent 64 values*/
				double values[4][4][4];
				int temp_c, temp_r, temp_rr;
				for (int rr = k - 1, ss = 0; rr <= k + 2; rr++, ss++){
					for (int r = j - 1, s = 0; r <= j + 2; r++, s++){
						for (int c = i - 1, t = 0; c <= i + 2; c++, t++){
							//todo: ??64?,????
							temp_c = min(max(Dtype(c), Dtype(0)), Dtype(width - 1));
							temp_r = min(max(Dtype(r), Dtype(0)), Dtype(height - 1));
							temp_rr = min(max(Dtype(rr), Dtype(0)), Dtype(length - 1));
							values[ss][s][t] = bottom_data[temp_rr*height*width + temp_r*width + temp_c];
							argmax_temp_data[ss * 16 + s * 4 + t] = temp_rr*height*width + temp_r*width + temp_c;
						}
					}
				}

				/*calc the coeff*/
				double u = wcenter - i;
				double v = hcenter - j;
				double vv = lcenter - k;
				double A[4], C[4], CC[4];
				for (int distance = 1, s = 0; distance >= -2; distance--, s++){
					A[s] = cubic_coeff_gpu(u + distance);
					C[s] = cubic_coeff_gpu(v + distance);
					CC[s] = cubic_coeff_gpu(vv + distance);
				}

				dfCubicValue = 0;
				for (int ss = 0; ss < 4; ss++) {
					for (int s = 0; s < 4; s++) {
						for (int t = 0; t < 4; t++) {
							dfCubicValue += values[ss][s][t] * A[t] * C[s] * CC[ss];
							w_temp_data[ss * 16 + s * 4 + t] = A[t] * C[s] * CC[ss];
						}
					}
				}
				if (dfCubicValue > maxValue || inter_index == 0) {
					maxValue = dfCubicValue;
					top_data[index] = dfCubicValue;
					for (int s = 0; s < 64; ++s) {
						w_data[64 * index + s] = w_temp_data[s];
						argmax_data[64 * index + s] = argmax_temp_data[s];
					}
				}
			}
		}
	}

	template <typename Dtype>
	void ROIAlignLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		const vector<Blob<Dtype>*>& top) {
		const Dtype* bottom_data = bottom[0]->gpu_data();
		const Dtype* bottom_rois = bottom[1]->gpu_data();
		Dtype* top_data = top[0]->mutable_gpu_data();
		int* argmax_data = bili_idx.mutable_gpu_data();
		Dtype* w_data = bili_w.mutable_gpu_data();
		int count = top[0]->count();
		int interpolate_times = is_multi_interpolate ? 8 : 1;
		// NOLINT_NEXT_LINE(whitespace/operators)
		if (bi_type == BiCubic) {
			ROICubicForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, bottom_data, spatial_scale_xy_, spatial_scale_z_, channels_, length_, height_, width_,
				pooled_length_, pooled_height_, pooled_width_, bottom_rois, interpolate_times, top_data, argmax_data, w_data);
		}
		else {
			ROIAlignForward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
				count, bottom_data, spatial_scale_xy_, spatial_scale_z_, channels_, length_, height_, width_,
				pooled_length_, pooled_height_, pooled_width_, bottom_rois, interpolate_times, top_data, argmax_data, w_data);
		}
		CUDA_POST_KERNEL_CHECK;
	}

	template <typename Dtype>
	__global__ void ROIAlignBackward(const int nthreads, const Dtype* top_diff,
		const int* argmax_data, const Dtype* w_data, const int num_rois, 
		const int channels, 
		const int length, const int height, const int width,
		const int pooled_length, const int pooled_height, const int pooled_width,
		const int w_num, Dtype* bottom_diff, const Dtype* bottom_rois) {
		CUDA_KERNEL_LOOP(index, nthreads) {
			// (n, c, pl, ph, pw) is an element in the pooled output
			//int pw = index % pooled_width;
			//int ph = (index / pooled_width) % pooled_height;
			//int pl = (index / pooled_width / pooled_height) % pooled_length;
			int c = (index / pooled_width / pooled_height / pooled_length) % channels;
			int n = index / pooled_width / pooled_height / pooled_length / channels;

			bottom_rois += n * 7;
			int roi_batch_ind = bottom_rois[0];

			for (int i = 0; i < w_num; ++i) {
				if (argmax_data[w_num * index + i] >= 0) {
					int offset_bottom = (roi_batch_ind * channels + c) * length * height
						* width + argmax_data[w_num * index + i];
					bottom_diff[offset_bottom] += top_diff[index] * w_data[w_num * index + i];
				}
			}
		}
	}

	template <typename Dtype>
	void ROIAlignLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
		const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
		if (!propagate_down[0]) {
			return;
		}
		const Dtype* bottom_rois = bottom[1]->gpu_data();
		const Dtype* top_diff = top[0]->gpu_diff();
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const int count = bottom[0]->count();
		caffe_gpu_set(count, Dtype(0.), bottom_diff);
		const int* argmax_data = bili_idx.gpu_data();
		const Dtype* w_data = bili_w.gpu_data();
		const int top_count = top[0]->count();
		int w_num = 8;
		if (bi_type == BiCubic) {
			w_num = 64;
		}
		// NOLINT_NEXT_LINE(whitespace/operators)
		ROIAlignBackward<Dtype> << <CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS >> >(
			top_count, top_diff, argmax_data, w_data, top[0]->num(), channels_,
			length_, height_, width_, pooled_length_, pooled_height_, pooled_width_, w_num, bottom_diff, bottom_rois);
		CUDA_POST_KERNEL_CHECK;
	}

	INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignLayer);

}  // namespace caffe