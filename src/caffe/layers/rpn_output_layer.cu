#include "hip/hip_runtime.h"
#include <iostream>  // NOLINT(readability/streams)
#include <fstream>  // NOLINT(readability/streams)
#include <string>
#include <utility>
#include <vector>
#include <stdio.h>

#include "caffe/layers/rpn_output_layer.hpp"
#include "caffe/util/benchmark.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/rng.hpp"

//#define OUTPUT_BOX_TO_FILE

using std::sqrt;
using std::floor;
using std::ceil;
using std::min;
using std::max;
using std::exp;
using std::log;

namespace caffe {

template <typename Dtype>
void RPNOutputLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {

	const int batch_size = bottom[0]->shape(0);
	const int class_num = bottom[0]->shape(1) / anchors_num_;
	const int shift_offset[3][7] = {
		0, -5, 5, 0, 0, 0, 0,
		0, 0, 0, -5, 5, 0, 0,
		0, 0, 0, 0, 0, -1, 1
	};
	//LOG(INFO) << "input image size(L x H x W): " << im_info[0] << "x" << im_info[1] << "x" << im_info[2];
	feat_size_[0] = bottom[0]->shape(4);
	feat_size_[1] = bottom[0]->shape(3);
	feat_size_[2] = bottom[0]->shape(2);
	//LOG(INFO) << "score map size(L x H x W): " << feat_size_[2] << "x" << feat_size_[1] << "x" << feat_size_[0];

	const RPNOutputParameter& rpn_output_param = this->layer_param_.rpn_output_param();
	const int feat_stride_xy = rpn_output_param.feat_stride_xy();
	const int feat_stride_z = rpn_output_param.feat_stride_z();
	const int all_anchors_num = feat_size_[2] * feat_size_[1] * feat_size_[0] * anchors_num_;
	double* pred_box = new double[batch_size * class_num * 7];
	memset(pred_box, 0, sizeof(double) * batch_size * class_num * 7);

	// generate all ref_boxes
	Dtype* all_anchors = new Dtype[all_anchors_num * 6];
#pragma omp parallel for
	for (int shift_z = 0; shift_z < feat_size_[2]; ++shift_z) {
		for (int shift_y = 0; shift_y < feat_size_[1]; ++shift_y) {
			for (int shift_x = 0; shift_x < feat_size_[0]; ++shift_x) {
				for (int m = 0; m < anchors_num_; ++m) {
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 0] =
						shift_x * feat_stride_xy + anchors_[m * 6 + 0];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 1] =
						shift_y * feat_stride_xy + anchors_[m * 6 + 1];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 2] =
						shift_z * feat_stride_z + anchors_[m * 6 + 2];

					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 3] =
						shift_x * feat_stride_xy + anchors_[m * 6 + 3];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 4] =
						shift_y * feat_stride_xy + anchors_[m * 6 + 4];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 5] =
						shift_z * feat_stride_z + anchors_[m * 6 + 5];
				}
			}
		}
	}

	for (int batch_id = 0; batch_id < batch_size; ++batch_id) {

		const Dtype* im_info = bottom[2]->cpu_data() + 19 * batch_id;
		const int image_size[3] = { (int)im_info[2], (int)im_info[1], (int)im_info[0] };
		const double image_spacing[3] = { im_info[5], im_info[4], im_info[3] };
		const double image_origin[3] = { im_info[8], im_info[7], im_info[6] };
		//const int image_origin_size[3] = { (int)im_info[11], (int)im_info[10], (int)im_info[9] };
		const double image_origin_spacing[3] = { im_info[14], im_info[13], im_info[12] };
		const double image_origin_origin[3] = { im_info[17], im_info[16], im_info[15] };
		const int image_scale = im_info[18];

		const Dtype* bbox_deltas_src = bottom[1]->cpu_data() + batch_id * all_anchors_num * 6;

		Dtype* bbox_deltas = new Dtype[all_anchors_num * 6];
		for (int m = 0; m < anchors_num_; ++m) {
			for (int l = 0; l < feat_size_[2]; ++l) {
				for (int h = 0; h < feat_size_[1]; ++h) {
					for (int w = 0; w < feat_size_[0]; ++w) {
						for (int n = 0; n < 6; ++n) {
							bbox_deltas[(((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m) * 6 + n] =
								bbox_deltas_src[(((m * 6 + n) * feat_size_[2] + l) * feat_size_[1] + h) * feat_size_[0] + w];
						}
					}
				}
			}
		}
		Dtype* proposal = new Dtype[all_anchors_num * 6];
		bool* keep_proposal = new bool[all_anchors_num];
		int keep_num = 0;
		double min_size = rpn_output_param.rpn_min_size() * image_scale;
//#pragma omp parallel for
		for (int i = 0; i < all_anchors_num; ++i) {
			// Convert anchors into proposals via bbox transformations
			double width = all_anchors[i * 6 + 3] - all_anchors[i * 6 + 0] + 1.0;
			double height = all_anchors[i * 6 + 4] - all_anchors[i * 6 + 1] + 1.0;
			double length = all_anchors[i * 6 + 5] - all_anchors[i * 6 + 2] + 1.0;
			double ctr_x = all_anchors[i * 6 + 0] + 0.5 * width;
			double ctr_y = all_anchors[i * 6 + 1] + 0.5 * height;
			double ctr_z = all_anchors[i * 6 + 2] + 0.5 * length;
			if (rpn_output_param.bbox_normalize())
			{
				for (int j = 0; j < 6; ++j)
				{
					bbox_deltas[i * 6 + j] = bbox_deltas[i * 6 + j] * rpn_output_param.bbox_normalize_std(j) + rpn_output_param.bbox_normalize_mean(j);
				}
			}
			double proposal_ctr_x = bbox_deltas[i * 6 + 0] * width + ctr_x;
			double proposal_ctr_y = bbox_deltas[i * 6 + 1] * height + ctr_y;
			double proposal_ctr_z = bbox_deltas[i * 6 + 2] * length + ctr_z;
			double proposal_w = exp(bbox_deltas[i * 6 + 3]) * width;
			double proposal_h = exp(bbox_deltas[i * 6 + 4]) * height;
			double proposal_l = exp(bbox_deltas[i * 6 + 5]) * length;
			proposal[i * 6 + 0] = proposal_ctr_x - 0.5 * proposal_w;
			proposal[i * 6 + 1] = proposal_ctr_y - 0.5 * proposal_h;
			proposal[i * 6 + 2] = proposal_ctr_z - 0.5 * proposal_l;
			proposal[i * 6 + 3] = proposal_ctr_x + 0.5 * proposal_w;
			proposal[i * 6 + 4] = proposal_ctr_y + 0.5 * proposal_h;
			proposal[i * 6 + 5] = proposal_ctr_z + 0.5 * proposal_l;

			// clip predicted boxes to image
			proposal[i * 6 + 0] = max(min(static_cast<double>(proposal[i * 6 + 0]), image_size[0] - 1.0), 0.0);
			proposal[i * 6 + 1] = max(min(static_cast<double>(proposal[i * 6 + 1]), image_size[1] - 1.0), 0.0);
			proposal[i * 6 + 2] = max(min(static_cast<double>(proposal[i * 6 + 2]), image_size[2] - 1.0), 0.0);
			proposal[i * 6 + 3] = max(min(static_cast<double>(proposal[i * 6 + 3]), image_size[0] - 1.0), 0.0);
			proposal[i * 6 + 4] = max(min(static_cast<double>(proposal[i * 6 + 4]), image_size[1] - 1.0), 0.0);
			proposal[i * 6 + 5] = max(min(static_cast<double>(proposal[i * 6 + 5]), image_size[2] - 1.0), 0.0);

			keep_proposal[i] = true;

			// only keep anchors inside the image
			keep_proposal[i] &= (
				all_anchors[i * 6 + 0] >= 0 &&
				all_anchors[i * 6 + 1] >= 0 &&
				all_anchors[i * 6 + 2] >= 0 &&
				all_anchors[i * 6 + 3] < im_info[2] &&
				all_anchors[i * 6 + 4] < im_info[1] &&
				all_anchors[i * 6 + 5] < im_info[0]);

			// remove predicted boxes with either height or width < threshold
			// (NOTE: convert rpn_min_size to input image scale stored in im_info[3])
			keep_proposal[i] &= (
				(proposal[i * 6 + 3] - proposal[i * 6 + 0] + 1) >= min_size &&
				(proposal[i * 6 + 4] - proposal[i * 6 + 1] + 1) >= min_size &&
				(proposal[i * 6 + 5] - proposal[i * 6 + 2] + 1) >= min_size);
			keep_num += keep_proposal[i];
		}
		delete[]bbox_deltas;

#ifdef OUTPUT_BOX_TO_FILE
		size_t pos = lines_[lines_id_].first.find_first_of('/');
		std::string file_name = lines_[lines_id_].first.substr(pos + 1);
		std::string output_file_name = rpn_output_param.roi_root_folder() + file_name + ".proposal.txt";
		FILE* file;
		file = fopen(output_file_name.c_str(), "w");
		if (file != NULL)
		{
#endif
			const Dtype* bg_score = bottom[0]->cpu_data() + batch_id * all_anchors_num * class_num;
			int* max_score_label = new int[all_anchors_num];
			memset(max_score_label, 0, sizeof(int) * all_anchors_num);
			Dtype* max_score = new Dtype[all_anchors_num];
			for (int m = 0; m < anchors_num_; ++m)
			{
				for (int l = 0; l < feat_size_[2]; ++l)
				{
					for (int h = 0; h < feat_size_[1]; ++h)
					{
						for (int w = 0; w < feat_size_[0]; ++w)
						{
							max_score[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m] =
								bg_score[((m * feat_size_[2] + l) * feat_size_[1] + h) * feat_size_[0] + w];
						}
					}
				}
			}
			for (int class_label = 1; class_label <= class_num - 1; ++class_label)
			{
				const Dtype* class_scores = bg_score + all_anchors_num * class_label;
				Dtype* score_buffer = new Dtype[all_anchors_num];
				for (int m = 0; m < anchors_num_; ++m)
				{
					for (int l = 0; l < feat_size_[2]; ++l)
					{
						for (int h = 0; h < feat_size_[1]; ++h)
						{
							for (int w = 0; w < feat_size_[0]; ++w)
							{
								score_buffer[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m] =
									class_scores[((m * feat_size_[2] + l) * feat_size_[1] + h) * feat_size_[0] + w];
							}
						}
					}
				}
				for (int i = 0; i < all_anchors_num; ++i)
				{
					if (keep_proposal[i]) {
						if (max_score[i] < score_buffer[i])
						{
							max_score[i] = score_buffer[i];
							max_score_label[i] = class_label;
						}
					}
				}
				delete[]score_buffer;
			}
			int* instance_num = new int[class_num];
			for (int i = 0; i < class_num; ++i)
			{
				instance_num[i] = 0;
			}
			for (int i = 0; i < all_anchors_num; ++i)
			{
				instance_num[max_score_label[i]]++;
			}
#ifdef OUTPUT_BOX_TO_FILE
			std::string all_box_file_name = rpn_output_param.roi_root_folder() + file_name + ".all.box.txt";
			FILE* all_box_file;
			all_box_file = fopen(all_box_file_name.c_str(), "w");
			if (all_box_file != NULL)
			{
#endif
				for (int class_label = 1; class_label < class_num; ++class_label)
				{
					int n = rpn_output_param.top_percent_proposal() * instance_num[class_label];
					if (n <= 0)
						n = 1;
					int* top_n_inds = new int[n];
					double* top_n_scores = new double[n];
					for (int i = 0; i < n; ++i)
					{
						top_n_inds[i] = -1;
						top_n_scores[i] = -1.0;
					}
					for (int i = 0; i < all_anchors_num; ++i)
					{
						if (max_score_label[i] == class_label)
						{
#ifdef OUTPUT_BOX_TO_FILE
							Dtype output_proposal[6];
							for (int j = 0; j < 2; ++j)
							{
								for (int k = 0; k < 3; ++k)
								{
									output_proposal[j * 3 + k] = ((proposal[i * 6 + j * 3 + k] + shift_offset[k][batch_id]) * image_spacing[k] + image_origin[k] - image_origin_origin[k]) / image_origin_spacing[k];
								}
							}

							fprintf(all_box_file, "%d %f %f %f %f %f %f %f\n",
								max_score_label[i],
								output_proposal[0],
								output_proposal[1],
								output_proposal[2],
								output_proposal[3],
								output_proposal[4],
								output_proposal[5],
								max_score[i]);
#endif

							if (max_score[i] > top_n_scores[0] && max_score[i] > rpn_output_param.fg_score_threshold())
							{
								top_n_inds[0] = i;
								top_n_scores[0] = max_score[i];
								for (int j = 0; j < n - 1; ++j)
								{
									if (top_n_scores[j] > top_n_scores[j + 1])
									{
										int tmp_ind = top_n_inds[j];
										top_n_inds[j] = top_n_inds[j + 1];
										top_n_inds[j + 1] = tmp_ind;
										double tmp_score = top_n_scores[j];
										top_n_scores[j] = top_n_scores[j + 1];
										top_n_scores[j + 1] = tmp_score;
									}
									else
									{
										break;
									}
								}
							}
						}
					}
					int post_n = n;
					for (int i = 0; i < n; ++i)
					{
						if (top_n_inds[i] < 0)
						{
							post_n--;
						}
						else
						{
							break;
						}
					}
					if (post_n > 0)
					{
						double avg_proposal[6] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };
						if (rpn_output_param.weighted_top_n_proposal())
						{
							double weight_sum = 0.0;
							for (int i = n - 1; i >= n - post_n; --i)
							{
								weight_sum += top_n_scores[i];
								for (int j = 0; j < 6; ++j)
								{
									avg_proposal[j] += top_n_scores[i] * proposal[top_n_inds[i] * 6 + j];
								}
							}
							for (int i = 0; i < 6; ++i)
							{
								avg_proposal[i] = avg_proposal[i] / weight_sum;
							}
						}
						else
						{
							for (int i = n - 1; i >= n - post_n; --i)
							{
								for (int j = 0; j < 6; ++j)
								{
									avg_proposal[j] += proposal[top_n_inds[i] * 6 + j];
								}
							}
							for (int i = 0; i < 6; ++i)
							{
								avg_proposal[i] = avg_proposal[i] / (double)post_n;
							}
						}
						for (int i = 0; i < 2; ++i)
						{
							for (int j = 0; j < 3; ++j)
							{
								avg_proposal[i * 3 + j] = ((avg_proposal[i * 3 + j] + shift_offset[j][batch_id]) * image_spacing[j] + image_origin[j] - image_origin_origin[j]) / image_origin_spacing[j];
							}
						}
#ifdef OUTPUT_BOX_TO_FILE
						fprintf(file, "%d %f %f %f %f %f %f %f\n",
							class_label,
							avg_proposal[0],
							avg_proposal[1],
							avg_proposal[2],
							avg_proposal[3],
							avg_proposal[4],
							avg_proposal[5],
							top_n_scores[n - 1]);
#endif
						pred_box[batch_id * class_num * 7 + class_label * 7 + 0] = avg_proposal[0];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 1] = avg_proposal[1];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 2] = avg_proposal[2];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 3] = avg_proposal[3];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 4] = avg_proposal[4];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 5] = avg_proposal[5];
						pred_box[batch_id * class_num * 7 + class_label * 7 + 6] = top_n_scores[n - 1];
					}
					delete[]top_n_inds;
					delete[]top_n_scores;
				}
#ifdef OUTPUT_BOX_TO_FILE
			}
			fclose(all_box_file);
#endif
			delete[]instance_num;
			delete[]max_score;
			delete[]max_score_label;
#ifdef OUTPUT_BOX_TO_FILE
		}
		fclose(file);
#endif
		delete[]proposal;
		delete[]keep_proposal;
	}

	delete[]all_anchors;

	size_t pos = lines_[lines_id_].first.find_first_of('/');
	std::string file_name = lines_[lines_id_].first.substr(pos + 1);
	std::string output_file_name = rpn_output_param.roi_root_folder() + file_name + ".pred.txt";
	FILE* output_file;
	output_file = fopen(output_file_name.c_str(), "w");
	if (output_file != NULL)
	{
		for (int class_id = 1; class_id < class_num; ++class_id)
		{
			int box_num = 0;
			double output_box[7] = { 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0 };
			for (int batch_id = 0; batch_id < batch_size; ++batch_id)
			{
				if (pred_box[batch_id * class_num * 7 + class_id * 7 + 6] > 0)
				{
					box_num++;
					for (int i = 0; i < 7; ++i)
					{
						output_box[i] += pred_box[batch_id * class_num * 7 + class_id * 7 + i];
					}
				}
			}
			if (box_num > 0)
			{
				for (int i = 0; i < 7; ++i)
				{
					output_box[i] = output_box[i] / box_num;
				}
				fprintf(output_file, "%d %f %f %f %f %f %f %f\n",
					class_id,
					output_box[0],
					output_box[1],
					output_box[2],
					output_box[3],
					output_box[4],
					output_box[5],
					output_box[6]);
			}
		}
	}
	fclose(output_file);

	delete[]pred_box;

	lines_id_ = (lines_id_ + 1) % lines_.size();
}

template <typename Dtype>
void RPNOutputLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(RPNOutputLayer);

}  // namespace caffe