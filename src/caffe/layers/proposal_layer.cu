#include "hip/hip_runtime.h"
#include <iostream>  // NOLINT(readability/streams)
#include <fstream>  // NOLINT(readability/streams)
#include <string>
#include <utility>
#include <vector>
#include <stdio.h>

#include "caffe/layers/proposal_layer.hpp"
#include "caffe/util/benchmark.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/util/rng.hpp"


using std::sqrt;
using std::floor;
using std::ceil;
using std::min;
using std::max;
using std::exp;
using std::log;

namespace caffe {

template <typename Dtype>
void ProposalLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
	const vector<Blob<Dtype>*>& top) {

	const ProposalParameter& proposal_param = this->layer_param_.proposal_param();

	const int batch_size = bottom[0]->shape(0);
	const int class_num = bottom[0]->shape(1) / anchors_num_;
	CHECK_EQ(class_num, 2) << "class_num != 2";
	const int proposal_num = proposal_param.proposal_num();

	feat_size_[0] = bottom[0]->shape(4);
	feat_size_[1] = bottom[0]->shape(3);
	feat_size_[2] = bottom[0]->shape(2);

	vector<int> top_shape(2);
	top_shape[0] = batch_size * proposal_num;
	top_shape[1] = 7;
	top[0]->Reshape(top_shape);

	// Enumerate all anchors
	const int feat_stride_xy = proposal_param.feat_stride_xy();
	const int feat_stride_z = proposal_param.feat_stride_z();
	const int all_anchors_num = feat_size_[2] * feat_size_[1] * feat_size_[0] * anchors_num_;
	Dtype* all_anchors = new Dtype[all_anchors_num * 6];
#pragma omp parallel for
	for (int shift_z = 0; shift_z < feat_size_[2]; ++shift_z) {
		for (int shift_y = 0; shift_y < feat_size_[1]; ++shift_y) {
			for (int shift_x = 0; shift_x < feat_size_[0]; ++shift_x) {
				for (int m = 0; m < anchors_num_; ++m) {
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 0] =
						shift_x * feat_stride_xy + anchors_[m * 6 + 0];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 1] =
						shift_y * feat_stride_xy + anchors_[m * 6 + 1];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 2] =
						shift_z * feat_stride_z + anchors_[m * 6 + 2];

					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 3] =
						shift_x * feat_stride_xy + anchors_[m * 6 + 3];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 4] =
						shift_y * feat_stride_xy + anchors_[m * 6 + 4];
					all_anchors[(((shift_z * feat_size_[1] + shift_y) * feat_size_[0] + shift_x) * anchors_num_ + m) * 6 + 5] =
						shift_z * feat_stride_z + anchors_[m * 6 + 5];
				}
			}
		}
	}

	for (int batch_id = 0; batch_id < batch_size; ++batch_id)
	{

		const Dtype* scores_src = bottom[0]->cpu_data() + batch_id * all_anchors_num * class_num + all_anchors_num;
		const Dtype* bbox_deltas_src = bottom[1]->cpu_data() + batch_id * all_anchors_num * 6;
		const Dtype* im_info = bottom[2]->cpu_data() + batch_id * 19;
		const int image_size[3] = { (int)im_info[2], (int)im_info[1], (int)im_info[0] };
		const double image_spacing[3] = { im_info[5], im_info[4], im_info[3] };
		const double image_origin[3] = { im_info[8], im_info[7], im_info[6] };
		//const int image_origin_size[3] = { (int)im_info[11], (int)im_info[10], (int)im_info[9] };
		const double image_origin_spacing[3] = { im_info[14], im_info[13], im_info[12] };
		const double image_origin_origin[3] = { im_info[17], im_info[16], im_info[15] };
		const int image_scale = im_info[18];
		//LOG(INFO) << "image size(L x H x W): " << im_info[0] << "x" << im_info[1] << "x" << im_info[2];
		//DLOG(INFO) << "scale: " << im_info[3];
		//LOG(INFO) << "score map size(L x H x W): " << feat_size_[2] << "x" << feat_size_[1] << "x" << feat_size_[0];

		Dtype* bbox_deltas = new Dtype[all_anchors_num * 6];
		Dtype* scores = new Dtype[all_anchors_num];
		//double score_max = -1000000;
		//double score_min = 1000000;
#pragma omp parallel for
		for (int m = 0; m < anchors_num_; ++m) {
			for (int l = 0; l < feat_size_[2]; ++l) {
				for (int h = 0; h < feat_size_[1]; ++h) {
					for (int w = 0; w < feat_size_[0]; ++w) {
						for (int n = 0; n < 6; ++n) {
							bbox_deltas[(((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m) * 6 + n] =
								bbox_deltas_src[(((m * 6 + n) * feat_size_[2] + l) * feat_size_[1] + h) * feat_size_[0] + w];
						}
						scores[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m] =
							scores_src[((m * feat_size_[2] + l) * feat_size_[1] + h) * feat_size_[0] + w];
						//if (score_min > scores[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m]) {
						//	score_min = scores[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m];
						//}
						//if (score_max < scores[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m]) {
						//	score_max = scores[((l * feat_size_[1] + h) * feat_size_[0] + w) * anchors_num_ + m];
						//}
					}
				}
			}
		}
		//LOG(INFO) << "max score = " << score_max;
		//LOG(INFO) << "min score = " << score_min;

		Dtype* proposal = new Dtype[all_anchors_num * 6];
		bool* proposal_keep = new bool[all_anchors_num];
		int keep_num = 0;
		double min_size = proposal_param.rpn_min_size() * image_scale;
//#pragma omp parallel for
		for (int i = 0; i < all_anchors_num; ++i) {
			// Convert anchors into proposals via bbox transformations
			double width = all_anchors[i * 6 + 3] - all_anchors[i * 6 + 0] + 1.0;
			double height = all_anchors[i * 6 + 4] - all_anchors[i * 6 + 1] + 1.0;
			double length = all_anchors[i * 6 + 5] - all_anchors[i * 6 + 2] + 1.0;
			double ctr_x = all_anchors[i * 6 + 0] + 0.5 * width;
			double ctr_y = all_anchors[i * 6 + 1] + 0.5 * height;
			double ctr_z = all_anchors[i * 6 + 2] + 0.5 * length;
			if (proposal_param.bbox_normalize())
			{
				for (int j = 0; j < 6; ++j)
				{
					bbox_deltas[i * 6 + j] = bbox_deltas[i * 6 + j] * proposal_param.bbox_normalize_std(j) + proposal_param.bbox_normalize_mean(j);
				}
			}
			double proposal_ctr_x = bbox_deltas[i * 6 + 0] * width + ctr_x;
			double proposal_ctr_y = bbox_deltas[i * 6 + 1] * height + ctr_y;
			double proposal_ctr_z = bbox_deltas[i * 6 + 2] * length + ctr_z;
			double proposal_w = exp(bbox_deltas[i * 6 + 3]) * width;
			double proposal_h = exp(bbox_deltas[i * 6 + 4]) * height;
			double proposal_l = exp(bbox_deltas[i * 6 + 5]) * length;
			proposal[i * 6 + 0] = proposal_ctr_x - 0.5 * proposal_w;
			proposal[i * 6 + 1] = proposal_ctr_y - 0.5 * proposal_h;
			proposal[i * 6 + 2] = proposal_ctr_z - 0.5 * proposal_l;
			proposal[i * 6 + 3] = proposal_ctr_x + 0.5 * proposal_w;
			proposal[i * 6 + 4] = proposal_ctr_y + 0.5 * proposal_h;
			proposal[i * 6 + 5] = proposal_ctr_z + 0.5 * proposal_l;

			// clip predicted boxes to image
			proposal[i * 6 + 0] = max(min(static_cast<double>(proposal[i * 6 + 0]), image_size[0] - 1.0), 0.0);
			proposal[i * 6 + 1] = max(min(static_cast<double>(proposal[i * 6 + 1]), image_size[1] - 1.0), 0.0);
			proposal[i * 6 + 2] = max(min(static_cast<double>(proposal[i * 6 + 2]), image_size[2] - 1.0), 0.0);
			proposal[i * 6 + 3] = max(min(static_cast<double>(proposal[i * 6 + 3]), image_size[0] - 1.0), 0.0);
			proposal[i * 6 + 4] = max(min(static_cast<double>(proposal[i * 6 + 4]), image_size[1] - 1.0), 0.0);
			proposal[i * 6 + 5] = max(min(static_cast<double>(proposal[i * 6 + 5]), image_size[2] - 1.0), 0.0);

			proposal_keep[i] = true;

			// only keep anchors inside the image
			proposal_keep[i] &= (
				all_anchors[i * 6 + 0] >= 0 &&
				all_anchors[i * 6 + 1] >= 0 &&
				all_anchors[i * 6 + 2] >= 0 &&
				all_anchors[i * 6 + 3] < im_info[2] &&
				all_anchors[i * 6 + 4] < im_info[1] &&
				all_anchors[i * 6 + 5] < im_info[0]);

			// remove predicted boxes with either height or width < threshold
			// (NOTE: convert rpn_min_size to input image scale stored in im_info[3])
			proposal_keep[i] &= (
				(proposal[i * 6 + 3] - proposal[i * 6 + 0] + 1) >= min_size &&
				(proposal[i * 6 + 4] - proposal[i * 6 + 1] + 1) >= min_size &&
				(proposal[i * 6 + 5] - proposal[i * 6 + 2] + 1) >= min_size);
			keep_num += proposal_keep[i];
		}
		delete[]bbox_deltas;

		Dtype* scores_tmp = new Dtype[keep_num];
		Dtype* proposal_tmp = new Dtype[keep_num * 6];
		keep_num = 0;
		for (int i = 0; i < all_anchors_num; ++i) {
			if (proposal_keep[i]) {
				scores_tmp[keep_num] = scores[i];
				for (int j = 0; j < 6; ++j) {
					proposal_tmp[keep_num * 6 + j] = proposal[i * 6 + j];
				}
				keep_num++;
			}
		}
		delete[]proposal_keep;
		delete[]scores;
		delete[]proposal;
		scores = scores_tmp;
		proposal = proposal_tmp;

		// sort all (proposal, score) pairs by score from highest to lowest
		// take top pre_nms_topN (e.g. 6000)
		int pre_nms_top_n = proposal_param.rpn_pre_nms_top_n();
		int pre_nms_num = min(keep_num, pre_nms_top_n);
		Dtype tmp;
		for (int i = keep_num - 1; i >= keep_num - pre_nms_num; --i) {
			for (int j = keep_num - 1; j >= keep_num - i; --j) {
				if (scores[j] > scores[j - 1]) {
					tmp = scores[j - 1];
					scores[j - 1] = scores[j];
					scores[j] = tmp;
					for (int k = 0; k < 6; ++k) {
						tmp = proposal[(j - 1) * 6 + k];
						proposal[(j - 1) * 6 + k] = proposal[j * 6 + k];
						proposal[j * 6 + k] = tmp;
					}
				}
			}
		}

		// nms
		const int post_nms_top_n = proposal_param.rpn_post_nms_top_n();
		const int post_nms_num = min(pre_nms_num, post_nms_top_n);
		const double nms_thresh = proposal_param.rpn_nms_threshold();
		bool* suppressed = new bool[pre_nms_num];
		int* nms_keep_indices = new int[pre_nms_num];
		int nms_keep_num = 0;
		memset(suppressed, 0, pre_nms_num * sizeof(bool));
		for (int i = 0; i < pre_nms_num; ++i) {
			if (suppressed[i]) {
				continue;
			}
			nms_keep_indices[nms_keep_num++] = i;
			if (nms_keep_num >= post_nms_num) {
				break;
			}
			double ix1 = proposal[i * 6 + 0];
			double iy1 = proposal[i * 6 + 1];
			double iz1 = proposal[i * 6 + 2];
			double ix2 = proposal[i * 6 + 3];
			double iy2 = proposal[i * 6 + 4];
			double iz2 = proposal[i * 6 + 5];
			double volume_i = (ix2 - ix1 + 1) * (iy2 - iy1 + 1) * (iz2 - iz1 + 1);
			for (int j = i + 1; j < pre_nms_num; ++j) {
				if (suppressed[j]) {
					continue;
				}
				double jx1 = proposal[j * 6 + 0];
				double jy1 = proposal[j * 6 + 1];
				double jz1 = proposal[j * 6 + 2];
				double jx2 = proposal[j * 6 + 3];
				double jy2 = proposal[j * 6 + 4];
				double jz2 = proposal[j * 6 + 5];
				double volume_j = (jx2 - jx1 + 1) * (jy2 - jy1 + 1) * (jz2 - jz1 + 1);
				double xx1 = max(ix1, jx1);
				double yy1 = max(iy1, jy1);
				double zz1 = max(iz1, jz1);
				double xx2 = min(ix2, jx2);
				double yy2 = min(iy2, jy2);
				double zz2 = min(iz2, jz2);
				double w = max(0.0, xx2 - xx1 + 1);
				double h = max(0.0, yy2 - yy1 + 1);
				double l = max(0.0, zz2 - zz1 + 1);
				double inter = w * h * l;
				double overlap = inter / (volume_i + volume_j - inter);
				if (overlap >= nms_thresh) {
					suppressed[j] = true;
				}
			}
		}
		delete[]suppressed;

		int bg_num = 0;
		Dtype* bg_scores = NULL;
		Dtype* bg_proposals = NULL;
		if (this->phase() == TRAIN) {
			int bg_num_pre = 0;
			Dtype* bg_scores_pre = new Dtype[keep_num];
			Dtype* bg_proposals_pre = new Dtype[keep_num * 6];
			for (int i = pre_nms_num; i < keep_num; ++i) {
				if (scores[i] < proposal_param.bg_score_threshold()) {
					bg_scores_pre[bg_num_pre] = scores[i];
					for (int j = 0; j < 6; ++j) {
						bg_proposals_pre[bg_num_pre * 6 + j] = proposal[i * 6 + j];
					}
					bg_num_pre++;
				}
			}
			if (bg_num_pre == 0) {
				for (int i = pre_nms_num; i < keep_num; ++i) {
					bg_scores_pre[bg_num_pre] = scores[i];
					for (int j = 0; j < 6; ++j) {
						bg_proposals_pre[bg_num_pre * 6 + j] = proposal[i * 6 + j];
					}
					bg_num_pre++;
				}
			}
			if (bg_num_pre > 0) {
				if (bg_num_pre <= proposal_num - nms_keep_num)
				{
					bg_num = bg_num_pre;
					bg_scores = new Dtype[bg_num_pre];
					bg_proposals = new Dtype[bg_num_pre * 6];
					for (int i = 0; i < bg_num_pre; ++i) {
						bg_scores[i] = bg_scores_pre[i];
						for (int j = 0; j < 6; ++j) {
							bg_proposals[i * 6 + j] = bg_proposals_pre[i * 6 + j];
						}
					}
				}
				else
				{
					int* bg_inds = new int[proposal_num - nms_keep_num];
					bg_scores = new Dtype[proposal_num - nms_keep_num];
					bg_proposals = new Dtype[(proposal_num - nms_keep_num) * 6];
					caffe::rng_t* rng = static_cast<caffe::rng_t*>(rng_->generator());
					while (bg_num < proposal_num - nms_keep_num && bg_num < bg_num_pre) {
						int i = (*rng)() % bg_num_pre;
						bool exists = false;
						for (int j = 0; j < bg_num; ++j) {
							if (bg_inds[j] == i) {
								exists = true;
								break;
							}
						}
						bg_inds[bg_num] = i;
						if (!exists) {
							bg_scores[bg_num] = bg_scores_pre[i];
							for (int j = 0; j < 6; ++j) {
								bg_proposals[bg_num * 6 + j] = bg_proposals_pre[i * 6 + j];
							}
							bg_num++;
						}
					}
					delete[]bg_inds;
				}
			}
			delete[]bg_scores_pre;
			delete[]bg_proposals_pre;
		}

		//FILE* output_file;
		//output_file = fopen("F:/proposal.txt", "w");
		//if (output_file != NULL)
		{
			Dtype* roi_data = top[0]->mutable_cpu_data() + batch_id * proposal_num * 7;
			for (int i = 0; i < proposal_num; ++i)
			{
				roi_data[i * 7 + 0] = -1;
			}
			for (int i = 0; i < nms_keep_num; ++i) {
				roi_data[i * 7 + 0] = batch_id;
				for (int j = 0; j < 6; ++j) {
					roi_data[i * 7 + 1 + j] = proposal[nms_keep_indices[i] * 6 + j];
				}
				//fprintf(output_file, "%d %f %f %f %f %f %f %f\n",
				//	1,					
				//	(roi_data[i * 7 + 1] * image_spacing[0] + image_origin[0] - image_origin_origin[0]) / image_origin_spacing[0],
				//	(roi_data[i * 7 + 2] * image_spacing[1] + image_origin[1] - image_origin_origin[1]) / image_origin_spacing[1],
				//	(roi_data[i * 7 + 3] * image_spacing[2] + image_origin[2] - image_origin_origin[2]) / image_origin_spacing[2],
				//	(roi_data[i * 7 + 4] * image_spacing[0] + image_origin[0] - image_origin_origin[0]) / image_origin_spacing[0],
				//	(roi_data[i * 7 + 5] * image_spacing[1] + image_origin[1] - image_origin_origin[1]) / image_origin_spacing[1],
				//	(roi_data[i * 7 + 6] * image_spacing[2] + image_origin[2] - image_origin_origin[2]) / image_origin_spacing[2],
				//	1);
			}
			for (int i = 0; i < bg_num; ++i) {
				roi_data[(i + nms_keep_num) * 7 + 0] = batch_id;
				for (int j = 0; j < 6; ++j) {
					roi_data[(i + nms_keep_num) * 7 + 1 + j] = bg_proposals[i * 6 + j];
				}
				//fprintf(output_file, "%d %f %f %f %f %f %f %f\n",
				//	1,
				//	(roi_data[(i + nms_keep_num) * 7 + 1] * image_spacing[0] + image_origin[0] - image_origin_origin[0]) / image_origin_spacing[0],
				//	(roi_data[(i + nms_keep_num) * 7 + 2] * image_spacing[1] + image_origin[1] - image_origin_origin[1]) / image_origin_spacing[1],
				//	(roi_data[(i + nms_keep_num) * 7 + 3] * image_spacing[2] + image_origin[2] - image_origin_origin[2]) / image_origin_spacing[2],
				//	(roi_data[(i + nms_keep_num) * 7 + 4] * image_spacing[0] + image_origin[0] - image_origin_origin[0]) / image_origin_spacing[0],
				//	(roi_data[(i + nms_keep_num) * 7 + 5] * image_spacing[1] + image_origin[1] - image_origin_origin[1]) / image_origin_spacing[1],
				//	(roi_data[(i + nms_keep_num) * 7 + 6] * image_spacing[2] + image_origin[2] - image_origin_origin[2]) / image_origin_spacing[2],
				//	1);
			}
		}
		//fclose(output_file);

		delete[]nms_keep_indices;
		delete[]scores;
		delete[]proposal;
		delete[]bg_scores;
		delete[]bg_proposals;
	}
	delete[]all_anchors;
}

template <typename Dtype>
void ProposalLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
}

INSTANTIATE_LAYER_GPU_FUNCS(ProposalLayer);

}  // namespace caffe