#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/dice_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DiceLossForwardGPU(const int nthreads, 
	      const int channel, const int spatial_dim, 
          const Dtype* prob_data, const Dtype* gt_data, 
		  Dtype* intersection_buffer, Dtype* union_buffer) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
	const int s = index % spatial_dim;
	Dtype max_score = -1;
	int max_score_cls = 0;
    for (int c = 0; c < channel; ++c) {
      if (max_score < prob_data[n * channel * spatial_dim + c * spatial_dim + s])
      {
        max_score = prob_data[n * channel * spatial_dim + c * spatial_dim + s];
        max_score_cls = c;
      }
    }
    for (int c = 0; c < channel; ++c) {
	  const Dtype label_value = (max_score_cls == c);
	  const Dtype gt_value = (gt_data[n * spatial_dim + s] == c);
	  intersection_buffer[n * channel * spatial_dim + c * spatial_dim + s] = 
		  label_value * gt_value;
	  union_buffer[n * channel * spatial_dim + c * spatial_dim + s] = 
		  label_value * label_value + gt_value * gt_value;
    }
  }
}

template <typename Dtype>
void DiceLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* prob_data = bottom[0]->gpu_data();
  const Dtype* gt_data = bottom[1]->gpu_data();
  const int count = bottom[0]->count();
  const int num = bottom[0]->shape(0);
  const int channel = bottom[0]->shape(1);
  const int spatial_dim = count / num / channel;
  const int nthreads = count / channel;
  Dtype* intersection_buffer = buffer_.mutable_gpu_data();
  Dtype* union_buffer = buffer_.mutable_gpu_diff();
  const Dtype* ones = ones_mask_.gpu_data();
  DiceLossForwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, channel, spatial_dim, 
	  prob_data, gt_data, intersection_buffer, union_buffer);
  Dtype* intersection_data = intersection_.mutable_cpu_data();
  Dtype* union_data = union_.mutable_cpu_data();
  for (int n = 0; n < num * channel; ++n) {
	caffe_gpu_dot(spatial_dim, intersection_buffer + n * spatial_dim, 
	    ones, intersection_data + n);
	caffe_gpu_dot(spatial_dim, union_buffer + n * spatial_dim,
		ones, union_data + n);
	union_data[n] += 0.00001;
  }
  Dtype* loss = top[0]->mutable_cpu_data();
  loss[0] = 0;
  for (int n = 0; n < num * channel; ++n) {
    loss[0] += 2 * intersection_data[n] / union_data[n];
  }
  loss[0] = loss[0] / num / channel;
  LOG(INFO) << "Average dice(GPU) = " << loss[0];
}

template <typename Dtype>
__global__ void DiceLossBackwardGPU(const int nthreads, 
	      const int channel, const int spatial_dim, 
	      const Dtype* prob_data, const Dtype* gt_data, 
	      const Dtype* intersection_data, const Dtype* union_data, 
		  Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
	const int s = index % spatial_dim;
    for (int c = 1; c < channel; ++c) {
      const Dtype prob_value = prob_data[n * channel * spatial_dim + c * spatial_dim + s];
	  const Dtype gt_value = (gt_data[n * spatial_dim + s] == c);
	  const Dtype union_value = union_data[n * channel + c];
	  const Dtype intersection_value = intersection_data[n * channel + c];
	  const Dtype diff =
		  2 * (gt_value * union_value / (union_value * union_value) -
		  2 * prob_value * intersection_value / (union_value * union_value));

	  bottom_diff[n * channel * spatial_dim + c * spatial_dim + s] -= diff;
	  bottom_diff[n * channel * spatial_dim + 0 * spatial_dim + s] += diff;
	}
  }
}

template <typename Dtype>
void DiceLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
	const int count = bottom[0]->count();
    const int num = bottom[0]->shape(0);
    const int channel = bottom[0]->shape(1);
    const int spatial_dim = count / num / channel;
	const int nthreads = count / channel;
	const Dtype* prob_data = bottom[0]->gpu_data();
    const Dtype* gt_data = bottom[1]->gpu_data();
	const Dtype* intersection_data = intersection_.gpu_data();
	const Dtype* union_data = union_.gpu_data();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	caffe_gpu_memset(count * sizeof(Dtype), 0, bottom_diff);
	DiceLossBackwardGPU<Dtype> << <CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, channel, spatial_dim, 
		prob_data, gt_data, intersection_data, union_data, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DiceLossLayer);

}  // namespace caffe
